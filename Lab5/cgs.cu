#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample implements a conjugate graident solver on GPU
 * using CUBLAS and CUSPARSE
 *
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

/* Using updated (v2) interfaces to cublas and cusparse */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

// Utilities and system includes
#include <helper_functions.h>  // helper for shared functions common to CUDA SDK samples
#include <hip/hip_runtime_api.h>       // helper function CUDA error checking and intialization

const char *sSDKname     = "conjugateGradient";

double mclock(){
	struct timeval tp;

	double sec,usec;
	gettimeofday( &tp, NULL );
	sec    = double( tp.tv_sec );
	usec   = double( tp.tv_usec )/1E6;
	return sec + usec;
}


#define dot_BS     32
#define kernel_BS  32

/* genTridiag: generate a random tridiagonal symmetric matrix */
void genTridiag(int *I, int *J, float *val, int N, int nz)
{
	double RAND_MAXi = 1e6;
	double val_r     = 12.345 * 1e5;

	I[0] = 0, J[0] = 0, J[1] = 1;
	val[0] = (float)val_r/RAND_MAXi + 10.0f;
	val[1] = (float)val_r/RAND_MAXi;
	int start;

	for (int i = 1; i < N; i++)
	{
		if (i > 1)
		{
			I[i] = I[i-1]+3;
		}
		else
		{
			I[1] = 2;
		}

		start = (i-1)*3 + 2;
		J[start] = i - 1;
		J[start+1] = i;

		if (i < N-1)
		{
			J[start+2] = i + 1;
		}

		val[start] = val[start-1];
		val[start+1] = (float)val_r/RAND_MAXi + 10.0f;

		if (i < N-1)
		{
			val[start+2] = (float)val_r/RAND_MAXi;
		}
	}

	I[N] = nz;
}


void cgs_basic(int argc, char **argv, int N, int M){

	//int M = 0, N = 0, 
	int nz = 0, *I = NULL, *J = NULL;
	float *val = NULL;
	const float tol = 1e-10f;
	const int max_iter = 1000;
	float *x;
	float *rhs;
	float a, b, na, r0, r1;
	int *d_col, *d_row;
	float *d_val, *d_x, dot;
	float *d_r, *d_p, *d_Ax;
	int k;
	float alpha, beta, alpham1;

	// This will pick the best possible CUDA capable device
	hipDeviceProp_t deviceProp;
	int devID = findCudaDevice(argc, (const char **)argv);

	if (devID < 0)
	{
		printf("exiting...\n");
		exit(EXIT_SUCCESS);
	}

	checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

	// Statistics about the GPU device
	printf("> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
			deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

	int version = (deviceProp.major * 0x10 + deviceProp.minor);

	if (version < 0x11)
	{
		printf("%s: requires a minimum CUDA compute 1.1 capability\n", sSDKname);
		hipDeviceReset();
		exit(EXIT_SUCCESS);
	}

	/* Generate a random tridiagonal symmetric matrix in CSR format */
	//M = N = 32*64;//10; //1048576;
	printf("M = %d, N = %d\n", M, N);
	nz = (N-2)*3 + 4;
	I = (int *)malloc(sizeof(int)*(N+1));
	J = (int *)malloc(sizeof(int)*nz);
	val = (float *)malloc(sizeof(float)*nz);
	genTridiag(I, J, val, N, nz);

	/*
	   for (int i = 0; i < nz; i++){
	   printf("%d\t", J[i]);
	   }
	   printf("\n");
	   for (int i = 0; i < nz; i++){
	   printf("%2f\t", val[i]);
	   }
	 */

	x = (float *)malloc(sizeof(float)*N);
	rhs = (float *)malloc(sizeof(float)*N);

	for (int i = 0; i < N; i++)
	{
		rhs[i] = 1.0;
		x[i] = 0.0;
	}

	/* Get handle to the CUBLAS context */
	hipblasHandle_t cublasHandle = 0;
	hipblasStatus_t hipblasStatus_t;
	hipblasStatus_t = hipblasCreate(&cublasHandle);

	checkCudaErrors(hipblasStatus_t);

	/* Get handle to the CUSPARSE context */
	hipsparseHandle_t cusparseHandle = 0;
	hipsparseStatus_t cusparseStatus;
	cusparseStatus = hipsparseCreate(&cusparseHandle);

	checkCudaErrors(cusparseStatus);

	hipsparseMatDescr_t descr = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descr);

	checkCudaErrors(cusparseStatus);

	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

	checkCudaErrors(hipMalloc((void **)&d_col, nz*sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&d_row, (N+1)*sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&d_val, nz*sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&d_x, N*sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&d_r, N*sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&d_p, N*sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&d_Ax, N*sizeof(float)));

	hipMemcpy(d_col, J, nz*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row, I, (N+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_val, val, nz*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_r, rhs, N*sizeof(float), hipMemcpyHostToDevice);

	alpha = 1.0;
	alpham1 = -1.0;
	beta = 0.0;
	r0 = 0.;


	double t_start = mclock();
	hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_x, &beta, d_Ax);

	hipblasSaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1);                                // PODMIEN FUNCKJE (I)
	hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);                        // PODMIEN FUNCKJE (II)

	k = 1;

	while (r1 > tol*tol && k <= max_iter)
	{
		if (k > 1)
		{
			b = r1 / r0;
			hipblasStatus_t = hipblasSscal(cublasHandle, N, &b, d_p, 1);                        // PODMIEN FUNCKJE (I)
			hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1);            // PODMIEN FUNCKJE (I)
		}
		else
		{
			hipblasStatus_t = hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1);                    // PODMIEN FUNCKJE (I)
		}

		hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_p, &beta, d_Ax); // PODMIEN FUNCKJE (III)
		hipblasStatus_t = hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot);                  // PODMIEN FUNCKJE (II)
		a = r1 / dot;

		hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &a, d_p, 1, d_x, 1);                    // PODMIEN FUNCKJE (I)
		na = -a;
		hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &na, d_Ax, 1, d_r, 1);                  // PODMIEN FUNCKJE (I)

		r0 = r1;
		hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);                    // PODMIEN FUNCKJE (II)
		hipDeviceSynchronize();
		printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
		k++;
	}
	printf("TIME OF CGS_BASIC = %f\n", mclock() - t_start);

	hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);

	float rsum, diff, err = 0.0;

	for (int i = 0; i < N; i++)
	{
		rsum = 0.0;

		for (int j = I[i]; j < I[i+1]; j++)
		{
			rsum += val[j]*x[J[j]];
		}

		diff = fabs(rsum - rhs[i]);

		if (diff > err)
		{
			err = diff;
		}
	}

	hipsparseDestroy(cusparseHandle);
	hipblasDestroy(cublasHandle);

	free(I);
	free(J);
	free(val);
	free(x);
	free(rhs);
	hipFree(d_col);
	hipFree(d_row);
	hipFree(d_val);
	hipFree(d_x);
	hipFree(d_r);
	hipFree(d_p);
	hipFree(d_Ax);

	hipDeviceReset();

	printf("Test Summary:  Error amount = %e\n", err);
	//exit((k <= max_iter) ? 0 : 1);


}

	__global__ void
vectorCopy(int elementsCount, const float *src, float *dest)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elementsCount)
	{
		dest[i] = src[i];
	}
}

	__global__ void
vectorAxpy(int elementsCount, const float *src, float *dest, float alpha)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elementsCount)
	{
		dest[i] += src[i] * alpha;
	}
}
	__global__ void
vectorScale(int elementsCount, float *vec, float alpha)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elementsCount)
	{
		vec[i] *= alpha;
	}
}

	__global__ void
sparseMatrixMultiplyByVec(int matrixSize, int nonZeroNumber, float* values, int* rowptr, int* colind, float* x, float* y)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i < matrixSize)
	{
		float sub = 0.0f;
		for(int j = rowptr[i] ; j < rowptr[i+1] ; j++) 
		{
			sub += values[j] * x[colind[j]];
		}
		y[i] = sub;
	}
}

	__global__ void
dotProduct(int vectorLength, float *vec1, float* vec2, float* result)
{
	const int TMP_SIZE = 256;
	__shared__ float tmp[TMP_SIZE];

	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < vectorLength)
	{
		tmp[threadIdx.x] = vec1[i] * vec2[i];
	}
	else 
	{
		tmp[threadIdx.x] = 0;
	}
	__syncthreads();
	if(threadIdx.x == 0) {
		for(int i = 1 ; i < TMP_SIZE ; i++) {
			tmp [0] += tmp[i];
		}
		atomicAdd(result, tmp[0]);
	}
}


void cgs_TODO(int argc, char **argv, int N, int M){

	//int M = 0, N = 0, 
	int nz = 0, *I = NULL, *J = NULL;
	float *val = NULL;
	const float tol = 1e-10f;
	const int max_iter = 1000;
	float *x;
	float *rhs;
	float a, b, na, r0, r1;
	int *d_col, *d_row;
	float *d_val, *d_x, dot;
	float *d_r, *d_p, *d_Ax;
	float* r1d;
	int k;
	float alpha, beta, alpham1;
	int threadsPerBlock = 256;
	int blocksPerGrid =(N + threadsPerBlock - 1) / threadsPerBlock;


	// This will pick the best possible CUDA capable device
	hipDeviceProp_t deviceProp;
	int devID = findCudaDevice(argc, (const char **)argv);

	if (devID < 0)
	{
		printf("exiting...\n");
		exit(EXIT_SUCCESS);
	}

	checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

	// Statistics about the GPU device
	printf("> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
			deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

	int version = (deviceProp.major * 0x10 + deviceProp.minor);

	if (version < 0x11)
	{
		printf("%s: requires a minimum CUDA compute 1.1 capability\n", sSDKname);
		hipDeviceReset();
		exit(EXIT_SUCCESS);
	}

	/* Generate a random tridiagonal symmetric matrix in CSR format */
	//M = N = 32*64;//10; //1048576;
	printf("M = %d, N = %d\n", M, N);
	nz = (N-2)*3 + 4;
	I = (int *)malloc(sizeof(int)*(N+1));
	J = (int *)malloc(sizeof(int)*nz);
	val = (float *)malloc(sizeof(float)*nz);
	genTridiag(I, J, val, N, nz);

	/*
	   for (int i = 0; i < nz; i++){
	   printf("%d\t", J[i]);
	   }
	   printf("\n");
	   for (int i = 0; i < nz; i++){
	   printf("%2f\t", val[i]);
	   }
	 */

	x = (float *)malloc(sizeof(float)*N);
	rhs = (float *)malloc(sizeof(float)*N);

	for (int i = 0; i < N; i++)
	{
		rhs[i] = 1.0;
		x[i] = 0.0;
	}

	/* Get handle to the CUBLAS context */
	hipblasHandle_t cublasHandle = 0;
	hipblasStatus_t hipblasStatus_t;
	hipblasStatus_t = hipblasCreate(&cublasHandle);

	checkCudaErrors(hipblasStatus_t);

	/* Get handle to the CUSPARSE context */
	hipsparseHandle_t cusparseHandle = 0;
	hipsparseStatus_t cusparseStatus;
	cusparseStatus = hipsparseCreate(&cusparseHandle);

	checkCudaErrors(cusparseStatus);

	hipsparseMatDescr_t descr = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descr);

	checkCudaErrors(cusparseStatus);

	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

	checkCudaErrors(hipMalloc((void **)&d_col, nz*sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&d_row, (N+1)*sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&d_val, nz*sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&d_x, N*sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&d_r, N*sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&d_p, N*sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&d_Ax, N*sizeof(float)));

	hipMemcpy(d_col, J, nz*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row, I, (N+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_val, val, nz*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_r, rhs, N*sizeof(float), hipMemcpyHostToDevice);

	alpha = 1.0;
	alpham1 = -1.0;
	// beta = 0.0;
	r0 = 0.;


	// sparse matrix vector product: d_Ax = A * d_x
	// hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_x, &beta, d_Ax);  // PODMIEN FUNCKJE (ZADANIE-I)
	sparseMatrixMultiplyByVec<<<blocksPerGrid, threadsPerBlock>>>(N, nz, d_val, d_row, d_col, d_x, d_Ax);  // PODMIEN FUNCKJE (ZADANIE-I)


	//azpy: d_r = d_r + alpham1 * d_Ax
	//hipblasSaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1);        			    // PODMIEN FUNCKJE (ZADANIE-I)
	vectorAxpy<<<blocksPerGrid, threadsPerBlock>>>(N, d_Ax, d_r, alpham1);
	//dot:  r1 = d_r * d_r
	// hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);                        // PODMIEN FUNCKJE (ZADANIE-III)
		
	hipMalloc(&r1d, sizeof(float));
	r1 = 0;
	hipMemcpy(r1d, &r1, sizeof(float), hipMemcpyHostToDevice);
	dotProduct<<<blocksPerGrid, threadsPerBlock>>>(N, d_r, d_r, r1d);                        // PODMIEN FUNCKJE (ZADANIE-III)	
	hipMemcpy(&r1, r1d, sizeof(float), hipMemcpyDeviceToHost);
	printf("r1: %f\n", r1);
	k = 1;

	while (r1 > tol*tol && k <= max_iter)
	{
		if (k > 1)
		{
			b = r1 / r0;
			//scal: d_p = b * d_p
			//hipblasStatus_t = hipblasSscal(cublasHandle, N, &b, d_p, 1);                        // PODMIEN FUNCKJE (ZADANIE-I)
			vectorScale<<<blocksPerGrid, threadsPerBlock>>>(N, d_p, b);                        // PODMIEN FUNCKJE (ZADANIE-I)
			
			//axpy:  d_p = d_p + alpha * d_r
			// hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1);            // PODMIEN FUNCKJE (ZADANIE-I)
			vectorAxpy<<<blocksPerGrid, threadsPerBlock>>>(N, d_r, d_p, alpha);            // PODMIEN FUNCKJE (ZADANIE-I)
		}
		else
		{
			//cpy: d_p = d_r
			vectorCopy<<<blocksPerGrid, threadsPerBlock >>>(N, d_r, d_p);                    				     // PODMIEN FUNCKJE (ZADANIE-I)
		}

		//sparse matrix-vector product: d_Ax = A * d_p
		// hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_p, &beta, d_Ax); // PODMIEN FUNCKJE (ZADANIE-II)
		sparseMatrixMultiplyByVec<<<blocksPerGrid, threadsPerBlock>>>(N, nz, d_val, d_row, d_col, d_p, d_Ax); // PODMIEN FUNCKJE (ZADANIE-II)
		
		// hipblasStatus_t = hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot);                  // PODMIEN FUNCKJE (ZADANIE-III)
		dot = 0.0;
		hipMemcpy(r1d, &dot, sizeof(float), hipMemcpyHostToDevice);
		dotProduct<<<blocksPerGrid, threadsPerBlock>>>(N, d_p, d_Ax, r1d);                        // PODMIEN FUNCKJE (ZADANIE-III)	
		hipMemcpy(&dot, r1d, sizeof(float), hipMemcpyDeviceToHost);
		
		a = r1 / dot;

		//axpy: d_x = d_x + a*d_p
		hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &a, d_p, 1, d_x, 1);                    // PODMIEN FUNCKJE (ZADANIE-I)
		na = -a;

		//axpy:  d_r = d_r + na * d_Ax
		hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &na, d_Ax, 1, d_r, 1);                  // PODMIEN FUNCKJE (ZADANIE-I)

		r0 = r1;

		//dot: r1 = d_r * d_r
		// hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);                    // PODMIEN FUNCKJE (ZADANIE-III)
		
		r1 = 0.0;
		hipMemcpy(r1d, &r1, sizeof(float), hipMemcpyHostToDevice);
		dotProduct<<<blocksPerGrid, threadsPerBlock>>>(N, d_r, d_r, r1d);                        // PODMIEN FUNCKJE (ZADANIE-III)	
		hipMemcpy(&r1, r1d, sizeof(float), hipMemcpyDeviceToHost);

		
		hipDeviceSynchronize();
		printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
		k++;
	}

	hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);

	float rsum, diff, err = 0.0;

	for (int i = 0; i < N; i++)
	{
		rsum = 0.0;

		for (int j = I[i]; j < I[i+1]; j++)
		{
			rsum += val[j]*x[J[j]];
		}

		diff = fabs(rsum - rhs[i]);

		if (diff > err)
		{
			err = diff;
		}
	}

	hipsparseDestroy(cusparseHandle);
	hipblasDestroy(cublasHandle);

	free(I);
	free(J);
	free(val);
	free(x);
	free(rhs);
	hipFree(d_col);
	hipFree(d_row);
	hipFree(d_val);
	hipFree(d_x);
	hipFree(d_r);
	hipFree(d_p);
	hipFree(d_Ax);

	hipDeviceReset();

	printf("Test Summary:  Error amount = %e\n", err);
	//exit((k <= max_iter) ? 0 : 1);


}







int main(int argc, char **argv)
{
	//int N = 1e6;//1 << 20;
	//int N = 256 * (1<<10)  -10 ; //1e6;//1 << 20;
	int N = 1e5;
	int M = N; 

	cgs_basic(argc, argv, N, M);

	cgs_TODO(argc, argv, N, M);
}
